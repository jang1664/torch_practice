#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>

template <typename scalar_t>
__global__ void add_cuda_forward_kernel(const scalar_t *__restrict__ x,
                                        const scalar_t *__restrict__ y, scalar_t *__restrict__ out,
                                        size_t N) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < N) {
    out[index] = x[index] + y[index];
  }
}

template <typename scalar_t>
__global__ void add_cuda_backward_kernel(const scalar_t *__restrict__ out_grad,
                                         scalar_t *__restrict__ x_grad,
                                         scalar_t *__restrict__ y_grad, const int N) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < N) {
    x_grad[index] = out_grad[index];
    y_grad[index] = out_grad[index];
  }
}

torch::Tensor add_cuda_forward(torch::Tensor x, torch::Tensor y) {

  const int n = x.size(0);
  const int threads = 1024;
  const dim3 blocks((n + threads - 1) / threads);

  torch::Tensor out = torch::zeros_like(x);

  AT_DISPATCH_FLOATING_TYPES(x.type(), "add_forward_cuda", ([&] {
                               add_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
                                   x.data<scalar_t>(), y.data<scalar_t>(), out.data<scalar_t>(), n);
                             }));

  return out;
}

std::vector<torch::Tensor> add_cuda_backward(torch::Tensor out_grad) {
  const int n = out_grad.size(0);
  const int threads = 1024;
  const dim3 blocks((n + threads - 1) / threads);

  torch::Tensor x_grad = torch::zeros_like(out_grad);
  torch::Tensor y_grad = torch::zeros_like(out_grad);

  AT_DISPATCH_FLOATING_TYPES(out_grad.type(), "add_backward_cuda", ([&] {
                               add_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
                                   out_grad.data<scalar_t>(), x_grad.data<scalar_t>(),
                                   y_grad.data<scalar_t>(), n);
                             }));
  return {x_grad, y_grad};
}